#include "hip/hip_runtime.h"
#include "phong.h"

//
// Pinhole/DOF camera implementation
//
rtDeclareVariable(float3, eye, , );
rtDeclareVariable(float3, U, , );
rtDeclareVariable(float3, V, , );
rtDeclareVariable(float3, W, , );

rtDeclareVariable(int, renderingDivisionLevel, , "Number of horizontal stripes");
rtDeclareVariable(int, myStripe, , "Current stripe");

rtBuffer<float4, 2> output_buffer;

rtDeclareVariable(int, AAlevel, , );
rtDeclareVariable(float, aperture_radius, , );
rtDeclareVariable(float, focal_length, , );
rtDeclareVariable(int, dof_samples, , );

RT_PROGRAM void dof_camera()
{
	float2 screen = make_float2(output_buffer.size() * AAlevel);
	float3 result = make_float3(0.0f);
	uint2 newLaunchIndex = make_uint2(launch_index.x, launch_index.y + myStripe * output_buffer.size().y / renderingDivisionLevel);
	unsigned int seed = (launch_index.x * 1920 + launch_index.y) * launch_index.x * launch_index.y;
	int count = 0;

	for(int i = 0; i < AAlevel; ++i)
		for(int j = 0; j < AAlevel; ++j)
		{
			float2 d = make_float2(AAlevel * newLaunchIndex.x + i, AAlevel * newLaunchIndex.y + j) / screen * 2.f - 1.f;
			float3 ray_direction = normalize(d.x * U + d.y * V + W);

			optix::Ray ray(eye, ray_direction, radiance_ray_type, scene_epsilon);

			PerRayData_radiance prd;
			prd.importance = 1.f;
			prd.depth = 0;

			rtTrace(top_object, ray, prd);
			result += prd.result;
			count++;

			for(int k = 1; k < dof_samples; ++k)
			{
				float2 d = make_float2(AAlevel * newLaunchIndex.x + i, AAlevel * newLaunchIndex.y + j) / screen * 2.f - 1.f;
				float3 ray_direction = normalize(d.x * U + d.y * V + W);

				optix::Ray ray(eye, ray_direction, radiance_ray_type, scene_epsilon);

				float2 circPoint = make_float2(rnd(seed) * 2.0f - 1.0f, rnd(seed) * 2.0f - 1.0f);
				float3 apertureOffset = make_float3(circPoint.x * aperture_radius, circPoint.y * aperture_radius, 0.0f);
				ray.origin += apertureOffset;
				ray.direction *= focal_length;
				ray.direction -= apertureOffset;
				ray.direction = normalize(ray.direction);

				PerRayData_radiance prd;
				prd.importance = 1.f;
				prd.depth = 0;

				rtTrace(top_object, ray, prd);
				
				if(fabs(fmaxf((result + prd.result) / (count + 1) - result / count)) < EPS)
					break;
				count++;
				result += prd.result;
			}
		}

	output_buffer[newLaunchIndex] = make_float4(result) / count;//(AAlevel * AAlevel * dof_samples);
}

//
// Enviormement map
//
rtTextureSampler<uchar4, 2, hipReadModeNormalizedFloat> envmap;
RT_PROGRAM void envmap_miss()
{
	float theta = atan2f(ray.direction.x, ray.direction.z);
	float phi = M_PIf * 0.5f - acosf(ray.direction.y);
	float u = (theta + M_PIf) * (0.5f * M_1_PIf);
	float v = 0.5f * (1.0f + sinf(phi));
	prd_radiance.result = make_float3(tex2D(envmap, u, v));
}

//
// Returns solid color for miss rays
//
rtDeclareVariable(float3, miss_color, , );
RT_PROGRAM void miss()
{
	prd_radiance.result = miss_color;
}

//
// Returns color from [miss_min, miss_max] lineary interpolated across ray inclination
//
rtDeclareVariable(float3, miss_min, , );
rtDeclareVariable(float3, miss_max, , );
RT_PROGRAM void gradient_miss()
{
	float phi = asinf(ray.direction.y);
	prd_radiance.result = 2.0f * phi / M_PIf * (miss_max - miss_min) + miss_min;
}

//
// Set pixel to solid color upon failure
//
rtDeclareVariable(float3, bad_color, , );

RT_PROGRAM void exception()
{
	uint2 newLaunchIndex = make_uint2(launch_index.x, launch_index.y + myStripe * output_buffer.size().y / renderingDivisionLevel);
	output_buffer[newLaunchIndex] = make_float4(bad_color, 1.0f);
}


