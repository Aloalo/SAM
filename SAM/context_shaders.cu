#include "hip/hip_runtime.h"
#include "phong.h"

//
// Pinhole camera implementation
//
rtDeclareVariable(uint2, launch_index, rtLaunchIndex, );
rtDeclareVariable(float3, eye, , );
rtDeclareVariable(float3, U, , );
rtDeclareVariable(float3, V, , );
rtDeclareVariable(float3, W, , );

rtDeclareVariable(int, renderingDivisionLevel, , );
rtDeclareVariable(int, myStripe, , );

rtBuffer<float4, 2> output_buffer;

RT_PROGRAM void pinhole_camera()
{
	float2 screen = make_float2(output_buffer.size());
	uint2 newLaunchIndex = make_uint2(launch_index.x, launch_index.y + myStripe * output_buffer.size().y / renderingDivisionLevel);
	float2 d = make_float2(newLaunchIndex) / screen * 2.f - 1.f;
	float3 ray_direction = normalize(d.x * U + d.y * V + W);

	optix::Ray ray(eye, ray_direction, radiance_ray_type, scene_epsilon);

	PerRayData_radiance prd;
	prd.importance = 1.f;
	prd.depth = 0;

	rtTrace(top_object, ray, prd);

	output_buffer[newLaunchIndex] = make_float4(prd.result);
}

//
// Pinhole camera implementation with MSAA
//

rtDeclareVariable(int, AAlevel, , );

RT_PROGRAM void pinhole_camera_AA()
{
	float2 screen = make_float2(output_buffer.size() * AAlevel);
	float4 result = make_float4(0.0f);
	uint2 newLaunchIndex = make_uint2(launch_index.x, launch_index.y + myStripe * output_buffer.size().y / renderingDivisionLevel);

	for(int i = 0; i < AAlevel; ++i)
		for(int j = 0; j < AAlevel; ++j)
		{
			float2 d = make_float2(AAlevel * newLaunchIndex.x + i, AAlevel * newLaunchIndex.y + j) / screen * 2.f - 1.f;
			float3 ray_direction = normalize(d.x * U + d.y * V + W);

			optix::Ray ray(eye, ray_direction, radiance_ray_type, scene_epsilon);

			PerRayData_radiance prd;
			prd.importance = 1.f;
			prd.depth = 0;

			rtTrace(top_object, ray, prd);
			result += make_float4(prd.result);
		}

	output_buffer[newLaunchIndex] = result / (AAlevel * AAlevel);
}

//
// Enviormement map
//
rtTextureSampler<float4, 2> envmap;
RT_PROGRAM void envmap_miss()
{
	float theta = atan2f(ray.direction.x, ray.direction.z);
	float phi = M_PIf * 0.5f - acosf(ray.direction.y);
	float u = (theta + M_PIf) * (0.5f * M_1_PIf);
	float v = 0.5f * (1.0f + sinf(phi));
	prd_radiance.result = make_float3(tex2D(envmap, u, v));
}

//
// Returns solid color for miss rays
//
rtDeclareVariable(float3, miss_color, , );
RT_PROGRAM void miss()
{
	prd_radiance.result = miss_color;
}

//
// Returns color from [miss_min, miss_max] lineary interpolated across ray inclination
//
rtDeclareVariable(float3, miss_min, , );
rtDeclareVariable(float3, miss_max, , );
RT_PROGRAM void gradient_miss()
{
	float phi = asinf(ray.direction.y);
	prd_radiance.result = 2.0f * phi / M_PIf * (miss_max - miss_min) + miss_min;
}

//
// Set pixel to solid color upon failure
//
rtDeclareVariable(float3, bad_color, , );

RT_PROGRAM void exception()
{
	uint2 newLaunchIndex = make_uint2(launch_index.x, launch_index.y + myStripe * output_buffer.size().y / renderingDivisionLevel);
	output_buffer[newLaunchIndex] = make_float4(bad_color, 1.0f);
}


