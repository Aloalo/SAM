#include "hip/hip_runtime.h"
#include <optix.h>
#include <optixu/optixu_math_namespace.h>
#include <optixu/optixu_matrix_namespace.h>
#include <optixu/optixu_aabb_namespace.h>

using namespace optix;

rtBuffer<float3> vertex_buffer;     
rtBuffer<float3> normal_buffer;
rtBuffer<float3> tangent_buffer;
rtBuffer<float2> texcoord_buffer;
rtBuffer<int3> index_buffer;
rtTextureSampler<float4, 2, hipReadModeElementType> normal_map;

rtDeclareVariable(float3, texcoord, attribute texcoord, ); 
rtDeclareVariable(float3, geometric_normal, attribute geometric_normal, ); 
rtDeclareVariable(float3, shading_normal, attribute shading_normal, ); 
rtDeclareVariable(optix::Ray, ray, rtCurrentRay, );

template<bool useNormalMap>
static __inline__ __device__ void mesh_intersect(int primIdx)
{
	int3 idx = index_buffer[primIdx];

	float3 p0 = vertex_buffer[idx.x];
	float3 p1 = vertex_buffer[idx.y];
	float3 p2 = vertex_buffer[idx.z];

	float3 n;
	float t, beta, gamma;
	if(intersect_triangle(ray, p0, p1, p2, n, t, beta, gamma))
	{
		if(rtPotentialIntersection(t))
		{
			if(texcoord_buffer.size() == 0 || idx.x < 0 || idx.y < 0 || idx.z < 0)
				texcoord = make_float3(0.0f, 0.0f, 0.0f);
			else 
			{
				float2 t0 = texcoord_buffer[idx.x];
				float2 t1 = texcoord_buffer[idx.y];
				float2 t2 = texcoord_buffer[idx.z];
				texcoord = make_float3(t1 * beta + t2 * gamma + t0 * (1.0f - beta - gamma));
			}
			
			geometric_normal = normalize(n);

			if(normal_buffer.size() == 0 || idx.x < 0 || idx.y < 0 || idx.z < 0)
				shading_normal = normalize(n);
			else
			{
				float3 n0 = normal_buffer[idx.x];
				float3 n1 = normal_buffer[idx.y];
				float3 n2 = normal_buffer[idx.z];
				shading_normal = normalize(n1 * beta + n2 * gamma + n0 * (1.0f - beta - gamma));
			}

			if(useNormalMap)
			{
				float3 shading_tangent = normalize(tangent_buffer[idx.y] * beta +
					tangent_buffer[idx.z] * gamma + tangent_buffer[idx.x] * (1.0f - beta - gamma));
				float3 normal = make_float3(tex2D(normal_map, texcoord.x, texcoord.y)) * 2.f - 1.f;
				float3 transformed_normal;
				transformed_normal.x = dot(shading_tangent, normal);
				transformed_normal.y = dot(cross(shading_tangent, shading_normal), normal);
				transformed_normal.z = dot(shading_normal, normal);
				shading_normal += transformed_normal;
				shading_normal = normalize(shading_normal);
			}

			rtReportIntersection(0);
		}
	}
}

RT_PROGRAM void mesh_intersect_normalmap(int primIdx)
{
	mesh_intersect<true>(primIdx);
}

RT_PROGRAM void mesh_intersect_no_normalmap(int primIdx)
{
	mesh_intersect<false>(primIdx);
}

RT_PROGRAM void mesh_bounds(int primIdx, float result[6])
{  
	const int3 idx = index_buffer[primIdx];

	const float3 v0 = vertex_buffer[idx.x];
	const float3 v1 = vertex_buffer[idx.y];
	const float3 v2 = vertex_buffer[idx.z];
	const float area = length(cross(v1 - v0, v2 - v0));

	optix::Aabb *aabb = (optix::Aabb*)result;

	if(area > 0.0f && !isinf(area))
	{
		aabb->m_min = fminf(fminf(v0, v1), v2);
		aabb->m_max = fmaxf(fmaxf(v0, v1), v2);
	}
	else
		aabb->invalidate();
}

