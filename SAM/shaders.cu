#include "hip/hip_runtime.h"
#include "phong.h"

rtDeclareVariable(float3, shading_normal, attribute shading_normal, ); 
rtDeclareVariable(float3, geometric_normal, attribute geometric_normal, ); 

//
// Pinhole camera implementation
//
rtDeclareVariable(uint2, launch_index, rtLaunchIndex, );
rtDeclareVariable(float3, eye, , );
rtDeclareVariable(float3, U, , );
rtDeclareVariable(float3, V, , );
rtDeclareVariable(float3, W, , );

rtBuffer<float4, 2> output_buffer;

RT_PROGRAM void pinhole_camera()
{
	float2 screen = make_float2(output_buffer.size());

	float2 d = make_float2(launch_index) / screen * 2.f - 1.f;
	float3 ray_origin = eye;
	float3 ray_direction = normalize(d.x * U + d.y * V + W);

	optix::Ray ray(ray_origin, ray_direction, radiance_ray_type, scene_epsilon);

	PerRayData_radiance prd;
	prd.importance = 1.f;
	prd.depth = 0;

	rtTrace(top_object, ray, prd);

	output_buffer[launch_index] = make_float4(prd.result);
}

//
// Returns solid color for miss rays
//
rtDeclareVariable(float3, miss_color, , );
RT_PROGRAM void miss()
{
	prd_radiance.result = miss_color;
}

//
// Returns color from [miss_min, miss_max] lineary interpolated across ray inclination
//
rtDeclareVariable(float3, miss_min, , );
rtDeclareVariable(float3, miss_max, , );
RT_PROGRAM void gradient_miss()
{
	float phi = asinf(ray.direction.y);
	prd_radiance.result = 2.0f * phi / pi * (miss_max - miss_min) + miss_min;
}

//
// Set pixel to solid color upon failure
//
rtDeclareVariable(float3, bad_color, , );

RT_PROGRAM void exception()
{
	output_buffer[launch_index] = make_float4(bad_color, 1.0f);
}

//
// Terminates and fully attenuates ray after any hit
//
RT_PROGRAM void any_hit_solid()
{
	phongShadowed();
}

//
//ADS phong shader with shadows
//
rtDeclareVariable(float3, Ka, , );
rtDeclareVariable(float3, Kd, , );
rtDeclareVariable(float3, Ks, , );
rtDeclareVariable(float, phong_exp, , );
rtDeclareVariable(float3, reflectivity, , );

RT_PROGRAM void closest_hit_phong()
{
	float3 world_geo_normal = normalize(rtTransformNormal(RT_OBJECT_TO_WORLD, geometric_normal));
	float3 world_shade_normal = normalize(rtTransformNormal(RT_OBJECT_TO_WORLD, shading_normal));
	float3 ffnormal =  faceforward(world_shade_normal, -ray.direction, world_geo_normal);

	phongShade(Ka, Kd, Ks, ffnormal, phong_exp, reflectivity);
}

//
// Transparent object shadows
//
rtDeclareVariable(float3, shadow_attenuation, , );

RT_PROGRAM void any_hit_shadow_glass()
{
	float3 world_normal = normalize(rtTransformNormal(RT_OBJECT_TO_WORLD, shading_normal));
	float nDi = fabs(dot(world_normal, ray.direction));

	prd_shadow.attenuation *= 1 - fresnel_schlick(nDi, 5.0f, 1.0f - shadow_attenuation, make_float3(1.0f));

	rtIgnoreIntersection();
}

//
// Glass shader
//
rtDeclareVariable(float3, cutoff_color, , );
rtDeclareVariable(float, fresnel_exponent, , );
rtDeclareVariable(float, fresnel_minimum, , );
rtDeclareVariable(float, fresnel_maximum, , );
rtDeclareVariable(float, refraction_index, , );
rtDeclareVariable(float3, refraction_color, , );
rtDeclareVariable(float3, reflection_color, , );
rtDeclareVariable(float3, extinction_constant, , );
rtDeclareVariable(int, use_internal_reflections, , );

RT_PROGRAM void closest_hit_glass()
{
	const float3 h = ray.origin + t_hit * ray.direction;
	const float3 n = normalize(rtTransformNormal(RT_OBJECT_TO_WORLD, shading_normal)); // normal
	const float3 i = ray.direction;// incident direction

	float reflection = 1.0f;
	float3 result = make_float3(0.0f);

	float3 beer_attenuation;
	if(dot(n, ray.direction) > 0)
		beer_attenuation = exp(extinction_constant * t_hit);
	else
		beer_attenuation = make_float3(1);

	bool inside = false;

	if(prd_radiance.depth < max_depth)
	{
		float3 t;
		if(refract(t, i, n, refraction_index))
		{
			// check for external or internal reflection
			float cos_theta = dot(i, n);
			if(cos_theta < 0.0f)
				cos_theta = -cos_theta;
			else
			{
				inside = true;
				cos_theta = dot(t, n);
			}

			reflection = fresnel_schlick(cos_theta, fresnel_exponent, fresnel_minimum, fresnel_maximum);

			float importance = prd_radiance.importance * (1.0f - reflection) * optix::luminance(refraction_color * beer_attenuation);
			if(importance > importance_cutoff)
			{
				optix::Ray ray(h, t, radiance_ray_type, scene_epsilon);
				PerRayData_radiance refr_prd;
				refr_prd.depth = prd_radiance.depth+1;
				refr_prd.importance = importance;

				rtTrace(top_object, ray, refr_prd);
				result += (1.0f - reflection) * refraction_color * refr_prd.result;
			}
			else
				result += (1.0f - reflection) * refraction_color * cutoff_color;
		}

		float3 r = reflect(i, n);

		float importance = prd_radiance.importance * reflection * optix::luminance(reflection_color * beer_attenuation);
		if(importance > importance_cutoff && (!inside || (inside && use_internal_reflections)))
		{
			optix::Ray ray(h, r, radiance_ray_type, scene_epsilon);
			PerRayData_radiance refl_prd;
			refl_prd.depth = prd_radiance.depth + 1;
			refl_prd.importance = importance;

			rtTrace(top_object, ray, refl_prd);
			result += reflection * reflection_color * refl_prd.result;
		}
		else
			result += reflection * reflection_color * cutoff_color;
	}

	result = result * beer_attenuation;
	prd_radiance.result = result;
}

rtTextureSampler<float4, 2> ambient_map;        
rtTextureSampler<float4, 2> diffuse_map;
rtTextureSampler<float4, 2> specular_map;

rtDeclareVariable(float3, texcoord, attribute texcoord, ); 

RT_PROGRAM void closest_hit_mesh()
{
	float3 direction = ray.direction;
	float3 world_shading_normal = normalize(rtTransformNormal(RT_OBJECT_TO_WORLD, shading_normal));
	float3 world_geometric_normal = normalize(rtTransformNormal(RT_OBJECT_TO_WORLD, geometric_normal));
	float3 ffnormal = faceforward(world_shading_normal, -direction, world_geometric_normal);
	float3 uv = texcoord;

	float3 black = make_float3(0.0f, 0.0f, 0.0f);
	
	/*float3 pKa = make_float3(tex2D(ambient_map, uv.x, uv.y)) * Ka;
	float3 pKd = make_float3(tex2D(diffuse_map, uv.x, uv.y)) * Kd;
	float3 pKs = make_float3(tex2D(specular_map, uv.x, uv.y)) * Ks;*/

	float3 pKa = make_float3(tex2D(ambient_map, uv.x, uv.y));
	float3 pKd = make_float3(tex2D(diffuse_map, uv.x, uv.y));
	float3 pKs = make_float3(tex2D(specular_map, uv.x, uv.y));

	phongShade(pKa, pKd, pKs, ffnormal, phong_exp, reflectivity);
}
