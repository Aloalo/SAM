#include "hip/hip_runtime.h"
#include "helper.h"

rtDeclareVariable(float3, shading_normal, attribute shading_normal, ); 
rtDeclareVariable(float3, geometric_normal, attribute geometric_normal, ); 

rtDeclareVariable(PerRayData_radiance, prd_radiance, rtPayload, );
rtDeclareVariable(PerRayData_shadow, prd_shadow,   rtPayload, );

rtDeclareVariable(float, t_hit, rtIntersectionDistance, );
rtDeclareVariable(optix::Ray, ray, rtCurrentRay, );
rtDeclareVariable(uint2, launch_index, rtLaunchIndex, );

rtDeclareVariable(unsigned int, radiance_ray_type, , );
rtDeclareVariable(unsigned int, shadow_ray_type, , );
rtDeclareVariable(float, scene_epsilon, , );
rtDeclareVariable(rtObject, top_object, , );

rtDeclareVariable(float3, ambient_light_color, , );

//
// Pinhole camera implementation
//
rtDeclareVariable(float3, eye, , );
rtDeclareVariable(float3, U, , );
rtDeclareVariable(float3, V, , );
rtDeclareVariable(float3, W, , );

rtDeclareVariable(float3, bad_color, , );
rtBuffer<uchar4, 2> output_buffer;

RT_PROGRAM void pinhole_camera()
{
	size_t2 screen = output_buffer.size();

	float2 d = make_float2(launch_index) / make_float2(screen) * 2.f - 1.f;
	float3 ray_origin = eye;
	float3 ray_direction = normalize(d.x * U + d.y * V + W);

	optix::Ray ray(ray_origin, ray_direction, radiance_ray_type, scene_epsilon);

	PerRayData_radiance prd;
	prd.importance = 1.f;
	prd.depth = 0;

	rtTrace(top_object, ray, prd);

	output_buffer[launch_index] = make_color(prd.result);
}

//
// Returns solid color for miss rays
//
rtDeclareVariable(float3, miss_color, , );
RT_PROGRAM void miss()
{
	prd_radiance.result = miss_color;
}

//
// Returns color from [miss_min, miss_max] lineary interpolated across ray inclination
//
rtDeclareVariable(float3, miss_min, , );
rtDeclareVariable(float3, miss_max, , );
RT_PROGRAM void gradient_miss()
{
	float phi = asinf(ray.direction.y);
	prd_radiance.result = 2.0f * phi / pi * (miss_max - miss_min) + miss_min;
}

//
// Terminates and fully attenuates ray after any hit
//
RT_PROGRAM void any_hit_solid()
{
	prd_shadow.attenuation = make_float3(0.0f);
	rtTerminateRay();
}

static __device__ inline float3 get_ffnormal()
{
	float3 world_geo_normal = normalize(rtTransformNormal(RT_OBJECT_TO_WORLD, geometric_normal));
	float3 world_shade_normal = normalize(rtTransformNormal(RT_OBJECT_TO_WORLD, shading_normal));
	return faceforward(world_shade_normal, -ray.direction, world_geo_normal);
}

//
//ADS phong shader with shadows
//
rtDeclareVariable(float3, Ka, , );
rtDeclareVariable(float3, Kd, , );
rtDeclareVariable(float3, Ks, , );
rtDeclareVariable(float,  phong_exp, , );
rtBuffer<BasicLight> lights; 

static __device__ inline float3 phong_and_shadows(const float3 &ffnormal, const float3 &hit_point, const float3 &local_Kd)
{
	float3 color = Ka * ambient_light_color;
	for(int i = 0; i < lights.size(); ++i)
	{
		BasicLight light = lights[i];
		float3 L = normalize(light.pos - hit_point);
		float nDl = dot(ffnormal, L);

		if(nDl > 0.0f)
		{
			PerRayData_shadow shadow_prd;
			shadow_prd.attenuation = make_float3(1.0f);
			float Ldist = length(light.pos - hit_point);
			optix::Ray shadow_ray(hit_point, L, shadow_ray_type, scene_epsilon, Ldist);
			rtTrace(top_object, shadow_ray, shadow_prd);

			if(fmaxf(shadow_prd.attenuation) > 0.0f)
			{
				float3 light_color = light.color * shadow_prd.attenuation;
				color += local_Kd * nDl * light_color;

				float3 H = normalize(L - ray.direction);
				float nDh = dot(ffnormal, H);
				if(nDh > 0)
					color += Ks * light.color * powf(nDh, phong_exp);
			}
		}
	}
	return color;
}


RT_PROGRAM void closest_hit_phong()
{
	float3 ffnormal = get_ffnormal();
	float3 hit_point = ray.origin + t_hit * ray.direction;

	prd_radiance.result = phong_and_shadows(ffnormal, hit_point, Kd);
}

//
//ADS phong shader with shadows and procedural tile texture
//

rtDeclareVariable(float3, tile_v0, , );
rtDeclareVariable(float3, tile_v1, , );
rtDeclareVariable(float3, crack_color, , );
rtDeclareVariable(float, crack_width, , );

RT_PROGRAM void closest_hit_phong_tile()
{
	float3 ffnormal = get_ffnormal();
	float3 hit_point = ray.origin + t_hit * ray.direction;

	float v0 = dot(tile_v0, hit_point);
	float v1 = dot(tile_v1, hit_point);
	v0 = v0 - floor(v0);
	v1 = v1 - floor(v1);

	float3 local_Kd;
	if(v0 > crack_width && v1 > crack_width )
		local_Kd = Kd;
	else
		local_Kd = crack_color;

	prd_radiance.result = phong_and_shadows(ffnormal, hit_point, local_Kd);
}

//
//ADS phong shader with shadows and reflections
//
rtDeclareVariable(float3, reflectivity, , );
rtDeclareVariable(float, importance_cutoff, , );
rtDeclareVariable(int, max_depth, , );

RT_PROGRAM void closest_hit_reflection()
{
	float3 ffnormal = get_ffnormal();
	float3 hit_point = ray.origin + t_hit * ray.direction;
	float3 color = phong_and_shadows(ffnormal, hit_point, Kd);
	
	float3 r = schlick(-dot(ffnormal, ray.direction), reflectivity);
	float importance = prd_radiance.importance * optix::luminance(r);

	//reflection ray
	if(importance > importance_cutoff && prd_radiance.depth < max_depth)
	{
		PerRayData_radiance refl_prd;
		refl_prd.importance = importance;
		refl_prd.depth = prd_radiance.depth+1;
		float3 R = reflect(ray.direction, ffnormal);
		optix::Ray refl_ray(hit_point, R, radiance_ray_type, scene_epsilon);
		rtTrace(top_object, refl_ray, refl_prd);
		color += r * refl_prd.result;
	}

	prd_radiance.result = color;
}


//
// Transparent object shadows
//
rtDeclareVariable(float3, shadow_attenuation, , );

RT_PROGRAM void any_hit_shadow_glass()
{
  float3 world_normal = normalize(rtTransformNormal(RT_OBJECT_TO_WORLD, shading_normal));
  float nDi = fabs(dot(world_normal, ray.direction));

  prd_shadow.attenuation *= 1 - fresnel_schlick(nDi, 5, 1 - shadow_attenuation, make_float3(1));

  rtIgnoreIntersection();
}

//
// Glass shader
//
rtDeclareVariable(float3, cutoff_color, , );
rtDeclareVariable(float, fresnel_exponent, , );
rtDeclareVariable(float, fresnel_minimum, , );
rtDeclareVariable(float, fresnel_maximum, , );
rtDeclareVariable(float, refraction_index, , );
rtDeclareVariable(float3, refraction_color, , );
rtDeclareVariable(float3, reflection_color, , );
rtDeclareVariable(float3, extinction_constant, , );

RT_PROGRAM void closest_hit_glass()
{
	const float3 h = ray.origin + t_hit * ray.direction;
	const float3 n = normalize(rtTransformNormal(RT_OBJECT_TO_WORLD, shading_normal)); // normal
	const float3 i = ray.direction;// incident direction

	float reflection = 1.0f;
	float3 result = make_float3(0.0f);

	float3 beer_attenuation;
	if(dot(n, ray.direction) > 0)
		beer_attenuation = exp(extinction_constant * t_hit);
	else
		beer_attenuation = make_float3(1);

	// refraction
	if(prd_radiance.depth < max_depth)
	{
		float3 t;
		if(refract(t, i, n, refraction_index))
		{
			// check for external or internal reflection
			float cos_theta = dot(i, n);
			if(cos_theta < 0.0f)
				cos_theta = -cos_theta;
			else
				cos_theta = dot(t, n);

			reflection = fresnel_schlick(cos_theta, fresnel_exponent, fresnel_minimum, fresnel_maximum);

			float importance = prd_radiance.importance * (1.0f - reflection) * optix::luminance(refraction_color * beer_attenuation);
			if(importance > importance_cutoff)
			{
				optix::Ray ray(h, t, radiance_ray_type, scene_epsilon);
				PerRayData_radiance refr_prd;
				refr_prd.depth = prd_radiance.depth+1;
				refr_prd.importance = importance;

				rtTrace(top_object, ray, refr_prd);
				result += (1.0f - reflection) * refraction_color * refr_prd.result;
			}
			else
				result += (1.0f - reflection) * refraction_color * cutoff_color;
		}
	}

	// reflection
	if(prd_radiance.depth < max_depth)
	{
		float3 r = reflect(i, n);

		float importance = prd_radiance.importance * reflection * optix::luminance(reflection_color * beer_attenuation);
		if(importance > importance_cutoff)
		{
			optix::Ray ray(h, r, radiance_ray_type, scene_epsilon);
			PerRayData_radiance refl_prd;
			refl_prd.depth = prd_radiance.depth + 1;
			refl_prd.importance = importance;

			rtTrace(top_object, ray, refl_prd);
			result += reflection * reflection_color * refl_prd.result;
		}
		else
			result += reflection * reflection_color * cutoff_color;
	}

	result = result * beer_attenuation;
	prd_radiance.result = result;
}


//
// Set pixel to solid color upon failure
//
RT_PROGRAM void exception()
{
	output_buffer[launch_index] = make_color(bad_color);
}
