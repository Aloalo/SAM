#include "hip/hip_runtime.h"
#include "helper.h"

rtDeclareVariable(float3, shading_normal, attribute shading_normal, ); 

rtDeclareVariable(PerRayData_radiance, prd_radiance, rtPayload, );

rtDeclareVariable(optix::Ray, ray, rtCurrentRay, );
rtDeclareVariable(uint2, launch_index, rtLaunchIndex, );

rtDeclareVariable(unsigned int, radiance_ray_type, , );
rtDeclareVariable(float, scene_epsilon, , );
rtDeclareVariable(rtObject, top_object, , );


//
// Pinhole camera implementation
//
rtDeclareVariable(float3, eye, , );
rtDeclareVariable(float3, U, , );
rtDeclareVariable(float3, V, , );
rtDeclareVariable(float3, W, , );
rtDeclareVariable(float, magicAngle, , );
rtDeclareVariable(float, aspectRatio, , );

rtDeclareVariable(float3, bad_color, , );
rtBuffer<uchar4, 2> output_buffer;

RT_PROGRAM void pinhole_camera()
{
	size_t2 screen = output_buffer.size();

	float2 d = make_float2(launch_index) / screen.x * 2.f - 1.f;
	float3 ray_origin = eye;
	float3 ray_direction = normalize(d.x * U + d.y * V + W);

	optix::Ray ray(ray_origin, ray_direction, radiance_ray_type, scene_epsilon);

	PerRayData_radiance prd;
	prd.importance = 1.f;
	prd.depth = 0;

	rtTrace(top_object, ray, prd);

	output_buffer[launch_index] = make_color(prd.result);
}

//
// Returns solid color for miss rays
//
rtDeclareVariable(float3, miss_color, , );
RT_PROGRAM void miss()
{
	prd_radiance.result = miss_color;
}


//
// Returns shading normal as the surface shading result
// 
RT_PROGRAM void closest_hit_radiance()
{
	prd_radiance.result = normalize(rtTransformNormal(RT_OBJECT_TO_WORLD, shading_normal)) * 0.5f + 0.5f;
}


//
// Set pixel to solid color upon failure
//
RT_PROGRAM void exception()
{
	output_buffer[launch_index] = make_color(bad_color);
}
